#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2024, NVIDIA CORPORATION.  All rights reserved.
 * Copyright (c) 2021, NAVER Corp.  Authored by CLOVA.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "tensorrt_llm/common/logger.h"
#include "tensorrt_llm/common/memoryUtils.h"
#include "tensorrt_llm/common/reduceKernelUtils.cuh"
#include "tensorrt_llm/kernels/decodingCommon.h"
#include "tensorrt_llm/kernels/samplingAirTopPKernels.h"
#include "tensorrt_llm/kernels/samplingTopKKernels.h"
#include "tensorrt_llm/kernels/samplingTopPKernels.h"
#include "tensorrt_llm/layers/topPSamplingLayer.h"

#include <algorithm>
#include <float.h>

using namespace tensorrt_llm::common;
using namespace tensorrt_llm::kernels;
using namespace tensorrt_llm::runtime;

namespace tensorrt_llm
{
namespace layers
{

static __global__ void setTopPRuntimeArgs(SizeType batchSize, SizeType topK, SizeType* topKs, SizeType topKsSize,
    float topP, float* topPs, SizeType topPsSize, bool* skipDecode, SizeType const* batchSlots, float* initialTopPBuf)
{
    /**
     * @brief Setup the runtime arguments for topp, broadcasting top_p to top_ps
              and top_k to top_ks.
     */

    auto index = static_cast<SizeType>(blockIdx.x * blockDim.x + threadIdx.x);
    for (SizeType bi = index; bi < batchSize; bi += static_cast<SizeType>(gridDim.x * blockDim.x))
    {
        auto const batchSlot = batchSlots != nullptr ? batchSlots[bi] : bi;
        auto k = topKsSize > 1 ? topKs[batchSlot] : topK;
        auto const p = topPsSize > 1 ? topPs[batchSlot] : topP;
        if (k == 0 && p == 0.0f)
        {
            // TensorRT-LLM's topp implementation does not support topp = 0.0f, but it
            // equivalent to greedy search. So, we set the topk = 1 as an alternative
            // solution.
            k = 1;
        }
        topKs[batchSlot] = k;
        topPs[batchSlot] = p;
        skipDecode[batchSlot] = k > 0;

        initialTopPBuf[batchSlot] = topPs[batchSlot];
    }
}

template <typename T>
TopPSamplingLayer<T>::TopPSamplingLayer(SizeType maxBatchSize, SizeType vocabSize, SizeType vocabSizePadded,
    hipStream_t stream, std::shared_ptr<IAllocator> allocator, hipDeviceProp_t* prop, bool isDeterministic,
    bool isAirTopP)
    : BaseSamplingLayer<T>(maxBatchSize, vocabSize, vocabSizePadded, stream, std::move(allocator), prop)
    , mIsDeterministic(isDeterministic)
    , mIsAirTopP(isAirTopP)
{
    TLLM_LOG_TRACE("%s start", __PRETTY_FUNCTION__);

    allocateBuffer(mMaxBatchSize);

    TLLM_LOG_TRACE("%s stop", __PRETTY_FUNCTION__);
}

template <typename T>
TopPSamplingLayer<T>::~TopPSamplingLayer()
{
    TLLM_LOG_TRACE("%s start", __PRETTY_FUNCTION__);

    freeBuffer();

    TLLM_LOG_TRACE("%s stop", __PRETTY_FUNCTION__);
}

template <typename T>
void TopPSamplingLayer<T>::allocateBuffer(SizeType batchSize)
{
    TLLM_LOG_TRACE("%s start", __PRETTY_FUNCTION__);

    if (mIsAirTopP == false)
    {
        mSamplingWorkspaceSize = getTopPWorkspaceSize<T>(batchSize, mVocabSizePadded);
    }
    else
    {
        mSamplingWorkspaceSize = getAirTopPWorkspaceSize<T>(batchSize, mVocabSizePadded, mIsDeterministic);
    }

    std::array<size_t, 11> deviceBufferSizes;
    deviceBufferSizes[0] = sizeof(TokenIdType) * batchSize * mVocabSizePadded;
    deviceBufferSizes[1] = sizeof(SizeType) * (batchSize + 1);
    deviceBufferSizes[2] = sizeof(SizeType) * (batchSize + 1);
    deviceBufferSizes[3] = sizeof(SizeType) * batchSize;
    deviceBufferSizes[4] = sizeof(float) * batchSize;
    deviceBufferSizes[5] = sizeof(float) * batchSize;
    deviceBufferSizes[6] = sizeof(float) * batchSize;
    deviceBufferSizes[7] = sizeof(float) * batchSize;
    deviceBufferSizes[8] = sizeof(TokenIdType) * batchSize;
    deviceBufferSizes[9] = sizeof(bool) * batchSize;
    deviceBufferSizes[10] = *std::max_element(&deviceBufferSizes[3], &deviceBufferSizes[9]);

    mTopPIdValsDevice = mAllocator->reMalloc(mTopPIdValsDevice, deviceBufferSizes[0], false);
    mTopPOffsetDevice = mAllocator->reMalloc(mTopPOffsetDevice, deviceBufferSizes[1], false);
    mBeginTopPOffsetDevice = mAllocator->reMalloc(mBeginTopPOffsetDevice, deviceBufferSizes[2], false);
    mRuntimeTopKDevice = mAllocator->reMalloc(mRuntimeTopKDevice, deviceBufferSizes[3], false);
    mRuntimeTopPDevice = mAllocator->reMalloc(mRuntimeTopPDevice, deviceBufferSizes[4], false);
    mInitialTopPDevice = mAllocator->reMalloc(mInitialTopPDevice, deviceBufferSizes[5], false);
    mTopPDecayDevice = mAllocator->reMalloc(mTopPDecayDevice, deviceBufferSizes[6], false);
    mTopPMinDevice = mAllocator->reMalloc(mTopPMinDevice, deviceBufferSizes[7], false);
    mTopPResetIdsDevice = mAllocator->reMalloc(mTopPResetIdsDevice, deviceBufferSizes[8], false);
    mSkipDecodeDevice = mAllocator->reMalloc(mSkipDecodeDevice, deviceBufferSizes[9], false);
    mSetupWorkspaceDevice = mAllocator->reMalloc(mSetupWorkspaceDevice, deviceBufferSizes[10], false);

    mSkipDecodeHost = static_cast<bool*>(std::realloc(mSkipDecodeHost, sizeof(bool) * batchSize));
    std::fill(mSkipDecodeHost, mSkipDecodeHost + batchSize, true);

    mAllocatedSize = std::accumulate(deviceBufferSizes.begin(), deviceBufferSizes.end(), 0);
    TLLM_LOG_DEBUG("topPSamplingLayer allocated %lu bytes on GPU", mAllocatedSize);

    TLLM_LOG_TRACE("%s stop", __PRETTY_FUNCTION__);
}

template <typename T>
void TopPSamplingLayer<T>::freeBuffer()
{
    TLLM_LOG_TRACE("%s start", __PRETTY_FUNCTION__);

    mAllocator->free((void**) (&mTopPIdValsDevice));
    mAllocator->free((void**) (&mTopPOffsetDevice));
    mAllocator->free((void**) (&mBeginTopPOffsetDevice));
    mAllocator->free((void**) (&mRuntimeTopKDevice));
    mAllocator->free((void**) (&mRuntimeTopPDevice));
    mAllocator->free((void**) (&mInitialTopPDevice));
    mAllocator->free((void**) (&mTopPDecayDevice));
    mAllocator->free((void**) (&mTopPMinDevice));
    mAllocator->free((void**) (&mTopPResetIdsDevice));
    mAllocator->free((void**) (&mSkipDecodeDevice));
    mAllocator->free((void**) (&mSetupWorkspaceDevice));
    std::free(mSkipDecodeHost);

    TLLM_LOG_TRACE("%s stop", __PRETTY_FUNCTION__);
}

template <typename T>
void TopPSamplingLayer<T>::setup(SizeType const batchSize, SizeType const* batchSlots, SetupParams const& setupParams)
{
    TLLM_LOG_TRACE("%s start", __PRETTY_FUNCTION__);

    SizeType const defaultTopK = 0;
    auto runtimeTopK = setupParams.runtime_top_k.value_or(std::vector<SizeType>{defaultTopK});
    auto runtimeTopP = setupParams.runtime_top_p.value_or(std::vector<float>{});

    auto const runtimeTopKSize = runtimeTopK.size();
    auto const runtimeTopPSize = runtimeTopP.size();

    auto const defaultTopPDecay{1.0f};
    auto decayVec = setupParams.top_p_decay.value_or(std::vector<float>(batchSize, defaultTopPDecay));

    auto const defaultTopPMin{1e-6f}; // prevent topp becoming 0.0
    auto topPMinVec = setupParams.top_p_min.value_or(std::vector<float>(batchSize, defaultTopPMin));

    SizeType const defaultTopPResetId{-1};
    auto topPResetIdsVec = setupParams.top_p_reset_ids.value_or(std::vector<SizeType>(batchSize, defaultTopPResetId));

    if (runtimeTopPSize == 0)
    {
        for (SizeType bi = 0; bi < static_cast<SizeType>(batchSize); ++bi)
        {
            auto bid = bi;
            if (batchSlots)
            {
                bid = batchSlots[bi];
            }
            mSkipDecodeHost[bid] = true;
        }
        cudaAutoCpy(mSkipDecodeDevice, mSkipDecodeHost, mMaxBatchSize, mStream);
        return;
    }

    for (auto& topP : runtimeTopP)
    {
        if (topP < 0.f || topP > 1.0f)
        {
            TLLM_LOG_WARNING("TopP (%f) is out of range ([0.0, 1.0f]). Clip to closest number.", topP);
            topP = std::clamp(topP, 0.f, 1.f);
        }
    }

    for (auto& decay : decayVec)
    {
        if (decay <= 0.f || decay > 1.0f)
        {
            TLLM_LOG_WARNING("Decay (%f) is out of range ([0.0, 1.0f]). Change to 1.0.", decay);
            decay = 1.0f;
        }
    }

    for (auto& topPMin : topPMinVec)
    {
        if (topPMin <= 0.f || topPMin > 1.0f)
        {
            TLLM_LOG_WARNING("TopP min (%f) is out of range ([0.0, 1.0f]). Change to 0.5.", topPMin);
            topPMin = 0.5f;
        }
    }

    auto const topK = runtimeTopK.at(0);
    auto const topP = runtimeTopP.at(0);

    if (runtimeTopKSize > 1)
    {
        TLLM_CHECK_WITH_INFO(static_cast<SizeType>(runtimeTopK.size()) == batchSize,
            fmtstr("runtimeTopK.size() (%lu) == batchSize (%d) is not satisfied!", runtimeTopK.size(), batchSize));
        cudaAutoCpy(reinterpret_cast<SizeType*>(mSetupWorkspaceDevice), runtimeTopK.data(), batchSize, mStream);
        invokeScatterDecodingParams(
            reinterpret_cast<SizeType*>(mSetupWorkspaceDevice), mRuntimeTopKDevice, batchSlots, batchSize, mStream);
    }
    if (runtimeTopPSize > 1)
    {
        TLLM_CHECK_WITH_INFO(static_cast<SizeType>(runtimeTopP.size()) == batchSize,
            fmtstr("runtime_top_p.size() (%lu) == batchSize (%d) is not satisfied!", runtimeTopP.size(), batchSize));
        cudaAutoCpy(reinterpret_cast<float*>(mSetupWorkspaceDevice), runtimeTopP.data(), batchSize, mStream);
        invokeScatterDecodingParams(
            reinterpret_cast<float*>(mSetupWorkspaceDevice), mRuntimeTopPDevice, batchSlots, batchSize, mStream);
    }

    auto fillBuffers
        = [this, &batchSize, &batchSlots](std::string name, auto const& vector, auto deviceTmpBuffer, auto deviceBuffer)
    {
        TLLM_CHECK_WITH_INFO(static_cast<SizeType>(vector.size()) == batchSize,
            fmtstr("%s.size() (%lu) == batchSize (%d) is not satisfied!", name.c_str(), vector.size(), batchSize));
        cudaAutoCpy(deviceTmpBuffer, vector.data(), batchSize, mStream);
        invokeScatterDecodingParams(deviceTmpBuffer, deviceBuffer, batchSlots, batchSize, mStream);
    };

    fillBuffers("top_p_decay", decayVec, reinterpret_cast<float*>(mSetupWorkspaceDevice), mTopPDecayDevice);

    fillBuffers("top_p_min", topPMinVec, reinterpret_cast<float*>(mSetupWorkspaceDevice), mTopPMinDevice);

    fillBuffers(
        "top_p_reset_ids", topPResetIdsVec, reinterpret_cast<TokenIdType*>(mSetupWorkspaceDevice), mTopPResetIdsDevice);

    {
        dim3 block(std::min(static_cast<SizeType>(batchSize), 256));
        dim3 grid(divUp(static_cast<SizeType>(batchSize), static_cast<SizeType>(block.x)));
        setTopPRuntimeArgs<<<grid, block, 0, mStream>>>(batchSize, topK, mRuntimeTopKDevice, runtimeTopKSize, topP,
            mRuntimeTopPDevice, runtimeTopPSize, mSkipDecodeDevice, batchSlots, mInitialTopPDevice);
        sync_check_cuda_error();
    }

    cudaAutoCpy(mSkipDecodeHost, mSkipDecodeDevice, mMaxBatchSize, mStream);
    std::vector<float> runtimeTopPs(mMaxBatchSize);
    cudaAutoCpy(runtimeTopPs.data(), mRuntimeTopPDevice, mMaxBatchSize, mStream);
    {
        auto maxTopP = 0.f;
        for (SizeType bi = 0; bi < static_cast<SizeType>(batchSize); ++bi)
        {
            auto const bid = batchSlots ? batchSlots[bi] : bi;
            maxTopP = std::max(maxTopP, runtimeTopPs[bid]);
        }
        mRuntimeMaxTopP = std::max(mRuntimeMaxTopP, maxTopP);
    }

    if (mIsAirTopP == true)
    {
        int smCnt = 0;
        if (mCudaDeviceProp)
        {
            smCnt = mCudaDeviceProp->multiProcessorCount;
        }
        if (smCnt <= 0)
        {
            int deviceId;
            check_cuda_error(hipGetDevice(&deviceId)); // Get the correct device id
            hipDeviceProp_t prop;
            check_cuda_error(hipGetDeviceProperties(&prop, deviceId));
            smCnt = prop.multiProcessorCount;
        }
        mAirTopPBlockNum = calcAirTopPBlockNum<T>(batchSize, (int) mVocabSizePadded, smCnt, mIsDeterministic);
    }

    TLLM_LOG_TRACE("%s stop", __PRETTY_FUNCTION__);
}

template <typename T>
void TopPSamplingLayer<T>::forward(DecodingOutputParams& outputs, ForwardParams& inputs)
{
    TLLM_LOG_TRACE("%s start", __PRETTY_FUNCTION__);

    auto const batchSize = inputs.logits.shape[0];

    // Probabilities must be already computed instead of logits
    auto probs = inputs.logits.template getPtr<T>();
    auto endIds = inputs.end_ids.template getPtr<TokenIdType const>();
    auto batchSlots = inputs.batch_slots ? inputs.batch_slots->template getPtr<SizeType const>() : nullptr;
    auto curandStatesDevice = inputs.curand_states;
    auto samplingWorkspaceDevice = inputs.sampling_workspace;

    TLLM_CHECK_WITH_INFO(curandStatesDevice, "No hiprand states provided");
    TLLM_CHECK_WITH_INFO(samplingWorkspaceDevice, "No sampling workspace provided");

    if (mIsAirTopP == false)
    {
        invokeTopPInitialize(
            mTopPIdValsDevice, mTopPOffsetDevice, mBeginTopPOffsetDevice, batchSize, mVocabSizePadded, mStream);
        sync_check_cuda_error();
    }

    FinishedState* finishedInput = (inputs.finished)
        ? reinterpret_cast<FinishedState*>(inputs.finished->template getPtr<FinishedState::UnderlyingType>())
        : nullptr;
    FinishedState* finishedOutput = (outputs.finished)
        ? reinterpret_cast<FinishedState*>(outputs.finished->template getPtr<FinishedState::UnderlyingType>())
        : nullptr;

    auto cumLogProbs
        = (outputs.cum_log_probs) ? outputs.cum_log_probs->template getPtr<float>() : static_cast<float*>(nullptr);
    auto outputLogProbs = (outputs.output_log_probs) ? outputs.output_log_probs->template getPtr<float>()
                                                     : static_cast<float*>(nullptr);
    auto sequenceLength = (outputs.sequence_length) ? outputs.sequence_length->template getPtr<SizeType>()
                                                    : static_cast<SizeType*>(nullptr);

    if (mIsAirTopP == false)
    {
        invokeBatchTopPSampling<T>(samplingWorkspaceDevice, outputs.output_ids_ptr.template getPtr<int*>(),
            sequenceLength, finishedInput, finishedOutput, cumLogProbs, outputLogProbs, probs, mTopPIdValsDevice,
            mTopPOffsetDevice, mBeginTopPOffsetDevice, curandStatesDevice, batchSize, mMaxBatchSize, mVocabSizePadded,
            endIds, mRuntimeMaxTopP, mRuntimeTopPDevice, mStream, mSkipDecodeDevice, batchSlots);
    }
    else
    {
        invokeBatchAirTopPSampling<T>(samplingWorkspaceDevice, outputs.output_ids_ptr.template getPtr<int*>(),
            sequenceLength, finishedInput, finishedOutput, cumLogProbs, outputLogProbs, probs, curandStatesDevice,
            batchSize, mMaxBatchSize, mVocabSizePadded, endIds, mRuntimeMaxTopP, mRuntimeTopPDevice, mStream,
            mAirTopPBlockNum, mSkipDecodeDevice, batchSlots, mIsDeterministic);
    }

    sync_check_cuda_error();
    invokeComputeToppDecay(mRuntimeTopPDevice, mInitialTopPDevice,
        outputs.output_ids_ptr.template getPtr<TokenIdType const*>(), mTopPDecayDevice, mTopPMinDevice,
        mTopPResetIdsDevice, sequenceLength, batchSlots, batchSize, mStream);
    sync_check_cuda_error();
    TLLM_LOG_TRACE("%s stop", __PRETTY_FUNCTION__);
}

template class TopPSamplingLayer<float>;
template class TopPSamplingLayer<half>;

} // namespace layers
} // namespace tensorrt_llm
