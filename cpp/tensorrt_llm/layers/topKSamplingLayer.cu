#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2024, NVIDIA CORPORATION.  All rights reserved.
 * Copyright (c) 2021, NAVER Corp.  Authored by CLOVA.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "tensorrt_llm/common/logger.h"
#include "tensorrt_llm/common/memoryUtils.h"
#include "tensorrt_llm/kernels/decodingCommon.h"
#include "tensorrt_llm/kernels/samplingTopKKernels.h"
#include "tensorrt_llm/kernels/samplingTopPKernels.h"
#include "tensorrt_llm/layers/topKSamplingLayer.h"
#include "tensorrt_llm/runtime/iTensor.h"

#include <algorithm>
#include <float.h>

using namespace tensorrt_llm::common;
using namespace tensorrt_llm::kernels;
using namespace tensorrt_llm::runtime;

namespace tensorrt_llm
{
namespace layers
{

template <int32_t TOP_K_MAX>
__global__ void setupTopKRuntimeArgs(SizeType batchSize, SizeType topK, SizeType* topKs, SizeType topKsSize, float topP,
    float* topPs, SizeType topPsSize, bool* skipDecode, SizeType const* batchSlots)
{
    auto const index = static_cast<SizeType>(blockIdx.x * blockDim.x + threadIdx.x);
    for (auto bi = index; bi < batchSize; bi += static_cast<SizeType>(gridDim.x * blockDim.x))
    {
        auto const batchSlot = batchSlots != nullptr ? batchSlots[bi] : bi;
        auto k = topKsSize > 1 ? topKs[batchSlot] : topK;
        auto p = topPsSize > 1 ? topPs[batchSlot] : topP;
        if (k == 0 && p == 0.0f)
        {
            // TensorRT-LLM's topp implementation does not support topp = 0.0f, but it
            // equivalent to greedy search. So, we set the topk = 1 as an alternative
            // solution.
            k = 1;
        }
        if (k > 0 && p == 0.0f)
        {
            // This case corresponds to the old topk sampling, which is equivalent to
            // the old topk_topp sampling with topp=1.0f. TopKSamplingLayer and
            // TopKTopPSamplingLayer are now merged by TopKSamplingLayer. Thus, we
            // replace the case topk>0 and topp=0.0f by topk>0 and topp=1.0f for the
            // compatibility.
            p = 1.0f;
        }
        // Clip k value. A topk sampling kernel supports up to TOP_K_MAX.
        topKs[batchSlot] = k;
        // Clip p value if it is out of range. range = [0.0, 1.0].
        topPs[batchSlot] = p;
        skipDecode[batchSlot] = k == 0;
    }
}

template <typename T>
TopKSamplingLayer<T>::TopKSamplingLayer(SizeType maxBatchSize, SizeType vocabSize, SizeType vocabSizePadded,
    hipStream_t stream, std::shared_ptr<IAllocator> allocator)
    : BaseSamplingLayer<T>(maxBatchSize, vocabSize, vocabSizePadded, stream, std::move(allocator), nullptr)
{
    TLLM_LOG_TRACE("%s start", __PRETTY_FUNCTION__);

    allocateBuffer(mMaxBatchSize);

    TLLM_LOG_TRACE("%s stop", __PRETTY_FUNCTION__);
}

template <typename T>
TopKSamplingLayer<T>::~TopKSamplingLayer()
{
    TLLM_LOG_TRACE("%s start", __PRETTY_FUNCTION__);

    freeBuffer();

    TLLM_LOG_TRACE("%s stop", __PRETTY_FUNCTION__);
}

template <typename T>
void TopKSamplingLayer<T>::allocateBuffer(SizeType const batchSize)
{
    TLLM_LOG_TRACE("%s start", __PRETTY_FUNCTION__);

    mSamplingWorkspaceSize = getTopKWorkspaceSize<T>(batchSize, 1, TOP_K_MAX, mVocabSizePadded);

    std::array<size_t, 4> deviceBufferSizes;
    deviceBufferSizes[0] = sizeof(SizeType) * batchSize;
    deviceBufferSizes[1] = sizeof(float) * batchSize;
    deviceBufferSizes[2] = sizeof(bool) * batchSize;
    deviceBufferSizes[3] = std::max(deviceBufferSizes[0], deviceBufferSizes[1]);

    mRuntimeTopKDevice = mAllocator->reMalloc(mRuntimeTopKDevice, deviceBufferSizes[0], false);
    mRuntimeTopPDevice = mAllocator->reMalloc(mRuntimeTopPDevice, deviceBufferSizes[1], false);
    mSkipDecodeDevice = mAllocator->reMalloc(mSkipDecodeDevice, deviceBufferSizes[2], false);
    mSetupWorkspaceDevice = mAllocator->reMalloc(mSetupWorkspaceDevice, deviceBufferSizes[3], false);

    mSkipDecodeHost = static_cast<bool*>(std::realloc(mSkipDecodeHost, sizeof(bool) * batchSize));

    mAllocatedSize = std::accumulate(deviceBufferSizes.begin(), deviceBufferSizes.end(), 0);
    TLLM_LOG_DEBUG("topKSamplingLayer allocated %lu bytes on GPU", mAllocatedSize);

    TLLM_LOG_TRACE("%s stop", __PRETTY_FUNCTION__);
}

template <typename T>
void TopKSamplingLayer<T>::freeBuffer()
{
    TLLM_LOG_TRACE("%s start", __PRETTY_FUNCTION__);

    mAllocator->free((void**) (&mRuntimeTopKDevice));
    mAllocator->free((void**) (&mRuntimeTopPDevice));
    mAllocator->free((void**) (&mSkipDecodeDevice));
    mAllocator->free((void**) (&mSetupWorkspaceDevice));
    std::free(mSkipDecodeHost);

    TLLM_LOG_TRACE("%s stop", __PRETTY_FUNCTION__);
}

template <typename T>
void TopKSamplingLayer<T>::setup(SizeType const batchSize, SizeType const* batchSlots, SetupParams const& setupParams)
{
    TLLM_LOG_TRACE("%s start", __PRETTY_FUNCTION__);

    SizeType constexpr defaultTopK = 0;
    auto runtimeTopK = setupParams.runtime_top_k.value_or(std::vector<SizeType>{defaultTopK});
    auto runtimeTopP = setupParams.runtime_top_p.value_or(std::vector<float>{});

    auto const runtimeTopKSize = runtimeTopK.size();
    auto const runtimeTopPSize = runtimeTopP.size();
    mNormalizeLogProbs = setupParams.normalize_log_probs.has_value() && setupParams.normalize_log_probs.value();

    for (auto& topP : runtimeTopP)
    {
        if (topP < 0.f || topP > 1.0f)
        {
            TLLM_LOG_WARNING("TopP (%f) is out of range ([0.0, 1.0f]). Clip to closest number.", topP);
            topP = std::clamp(topP, 0.f, 1.f);
        }
    }
    for (auto& topK : runtimeTopK)
    {
        if (topK > TOP_K_MAX)
        {
            TLLM_LOG_WARNING(
                "TopK (%d) is larger than max supported number (%d). Clip to max supported number.", topK, TOP_K_MAX);
            topK = TOP_K_MAX;
        }
    }

    auto const topK = *std::max_element(std::begin(runtimeTopK), std::end(runtimeTopK));
    auto const topP = (runtimeTopPSize == 0) ? 0.0f : runtimeTopP.front();

    if (runtimeTopKSize > 1)
    {
        TLLM_CHECK_WITH_INFO(runtimeTopK.size() == batchSize,
            fmtstr("runtimeTopK.size() (%lu) == batchSize (%d) is not satisfied!", runtimeTopK.size(), batchSize));
        cudaAutoCpy(
            reinterpret_cast<runtime::SizeType*>(mSetupWorkspaceDevice), runtimeTopK.data(), batchSize, mStream);
        invokeScatterDecodingParams(reinterpret_cast<runtime::SizeType*>(mSetupWorkspaceDevice), mRuntimeTopKDevice,
            batchSlots, batchSize, mStream);
    }
    if (runtimeTopPSize > 1)
    {
        TLLM_CHECK_WITH_INFO(runtimeTopP.size() == batchSize,
            fmtstr("runtimeTopP.size() (%lu) == batchSize (%d) is not satisfied!", runtimeTopP.size(), batchSize));
        cudaAutoCpy(reinterpret_cast<float*>(mSetupWorkspaceDevice), runtimeTopP.data(), batchSize, mStream);
        invokeScatterDecodingParams(
            reinterpret_cast<float*>(mSetupWorkspaceDevice), mRuntimeTopPDevice, batchSlots, batchSize, mStream);
    }

    {
        dim3 block(std::min(static_cast<uint32_t>(batchSize), 256u));
        dim3 grid(divUp(static_cast<uint32_t>(batchSize), block.x));
        // support topK up to TOP_K_MAX.
        setupTopKRuntimeArgs<TOP_K_MAX><<<grid, block, 0, mStream>>>(batchSize, topK, mRuntimeTopKDevice,
            runtimeTopKSize, topP, mRuntimeTopPDevice, runtimeTopPSize, mSkipDecodeDevice, batchSlots);
    }

    cudaAutoCpy(mSkipDecodeHost, mSkipDecodeDevice, mMaxBatchSize, mStream);
    std::vector<SizeType> runtimeTopKs(mMaxBatchSize);
    cudaAutoCpy(runtimeTopKs.data(), mRuntimeTopKDevice, mMaxBatchSize, mStream);
    {
        runtime::SizeType maxTopK = 0;
        for (SizeType bi = 0; bi < static_cast<SizeType>(batchSize); ++bi)
        {
            auto bid = bi;
            if (batchSlots)
            {
                bid = batchSlots[bi];
            }
            maxTopK = std::max(maxTopK, runtimeTopKs[bid]);
        }
        mRuntimeMaxTopK = std::max(mRuntimeMaxTopK, maxTopK);
    }

    TLLM_LOG_TRACE("%s stop", __PRETTY_FUNCTION__);
}

template <typename T>
void TopKSamplingLayer<T>::forward(DecodingOutputParams& outputs, ForwardParams& inputs)
{
    TLLM_LOG_TRACE("%s start", __PRETTY_FUNCTION__);

    auto const batchSize = inputs.logits.shape[0];

    auto logits = inputs.logits.template getPtr<T>();
    auto endIds = inputs.end_ids.template getPtr<TokenIdType const>();
    auto batchSlots = inputs.batch_slots ? inputs.batch_slots->template getPtr<SizeType const>() : nullptr;
    auto curandStatesDevice = inputs.curand_states;
    auto samplingWorkspaceDevice = inputs.sampling_workspace;
    auto const probsComputed = inputs.probs_computed;

    TLLM_CHECK_WITH_INFO(curandStatesDevice, "No hiprand states provided");
    TLLM_CHECK_WITH_INFO(samplingWorkspaceDevice, "No sampling workspace provided");

    FinishedState* finishedInput = (inputs.finished)
        ? reinterpret_cast<FinishedState*>(inputs.finished->template getPtr<FinishedState::UnderlyingType>())
        : nullptr;
    FinishedState* finishedOutput = (outputs.finished)
        ? reinterpret_cast<FinishedState*>(outputs.finished->template getPtr<FinishedState::UnderlyingType>())
        : nullptr;

    auto cumLogProbs
        = (outputs.cum_log_probs) ? outputs.cum_log_probs->template getPtr<float>() : static_cast<float*>(nullptr);
    auto outputLogProbs = (outputs.output_log_probs) ? outputs.output_log_probs->template getPtr<float>()
                                                     : static_cast<float*>(nullptr);
    auto sequenceLength = (outputs.sequence_length) ? outputs.sequence_length->template getPtr<SizeType>()
                                                    : static_cast<SizeType*>(nullptr);

    invokeBatchTopKSampling(samplingWorkspaceDevice, logits, static_cast<T const* const*>(nullptr),
        outputs.output_ids_ptr.template getPtr<TokenIdType*>(), /* outputIds */ nullptr, sequenceLength, finishedInput,
        finishedOutput, cumLogProbs, outputLogProbs, curandStatesDevice, static_cast<SizeType>(mRuntimeMaxTopK),
        static_cast<SizeType*>(mRuntimeTopKDevice), 1.0f, mRuntimeTopPDevice, mVocabSizePadded, endIds, batchSlots,
        mStream, batchSize, mMaxBatchSize, /* tokens per step */ nullptr, /* max tokens per step */ 1,
        /* maxSeqLen ignored as outputIds is nullptr */ 0, mSkipDecodeDevice, mNormalizeLogProbs, probsComputed,
        /* return all Top-K*/ false);
    sync_check_cuda_error();

    TLLM_LOG_TRACE("%s stop", __PRETTY_FUNCTION__);
}

template class TopKSamplingLayer<float>;
template class TopKSamplingLayer<half>;

} // namespace layers
} // namespace tensorrt_llm
